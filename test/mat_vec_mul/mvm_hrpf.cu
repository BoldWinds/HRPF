#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <string>
#include <omp.h>
#include "algorithm/parallel_for_zero/parallel_for_zb.h"
#include "tool/initializer.h"
#include "framework/framework.h"
#include "tool/helper.h"

static int length;
struct UserData_t : public Basedata_t{
public:
    UserData_t(std::vector<Matrix*>m_bf, std::vector<ArrayList*> buf
        ) : m_buffer(m_bf), v_buffer(buf){
        }

public:
    std::vector<Matrix*> m_buffer;
    std::vector<ArrayList*> v_buffer;
};

void cfor_func(Basedata_t* data){
    auto d = (loopData_t*)data;
    auto a = ((UserData_t*)(d->buffer))->m_buffer[0]->get_cdata();
    auto b = ((UserData_t*)(d->buffer))->v_buffer[0]->get_cdata();
    auto c = ((UserData_t*)(d->buffer))->v_buffer[1]->get_cdata();

    size_t lda = ((UserData_t*)(d->buffer))->m_buffer[0]->get_ld();
    // size_t ldb = d->v_buffer[0]->get_ld();
    // size_t ldc = d->v_buffer[1]->get_ld();
    size_t s_i = d->start;
    size_t e_i = d->end;
    size_t s_j = 0;
    size_t e_j = length;
    // std::cout << s_i << s_j << e_i << e_j << std::endl;
    #pragma omp parallel for num_threads(16)
    for(int i = s_i; i < e_i; ++i){
        double loc = 0;
        for(int j = s_j; j < e_j; ++j) {
            loc += a[i + j * lda] * b[j];
            // std::cout << a[i + j * lda] << std::endl;
        }
        c[i] = loc;
    }
}

__global__ void kernel_2DMv(size_t s_i, size_t e_i, size_t s_j, size_t e_j,
    size_t lda, size_t ldb, size_t ldc,
    size_t chunk, double* a, double* b, double* c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int start_i = s_i + tid * chunk;
    int end_i = start_i + chunk < e_i ? start_i + chunk : e_i;

    for(int i = start_i; i < end_i; ++i){
        double loc = 0.0;
        for(int j = s_j; j < e_j; ++j) {
            loc += a[i + j * lda] * b[j];
        }
        c[i] = loc;
    }
}

void gfor_func(Basedata_t* data){
    auto d = (loopData_t*)data;
    auto a = ((UserData_t*)(d->buffer))->m_buffer[0]->get_gdata();
    auto b = ((UserData_t*)(d->buffer))->v_buffer[0]->get_gdata();
    auto c = ((UserData_t*)(d->buffer))->v_buffer[1]->get_gdata();

    size_t lda = ((UserData_t*)(d->buffer))->m_buffer[0]->get_ld();
    size_t s_i = d->start;
    size_t e_i = d->end;
    size_t s_j = 0;
    size_t e_j = length;

    int blocks_required = 1;
    int threads_per_block = 1024;
    int chunk_size = 1;
    int size = e_i - s_i;
    if(size % (threads_per_block * chunk_size)) {
        blocks_required = size / (threads_per_block * chunk_size) + 1;
    }
    else {
        blocks_required = size / (threads_per_block * chunk_size);
    }
    hipStream_t stream_ = stream();
    kernel_2DMv<<<blocks_required, threads_per_block, 0, stream_>>>(s_i, e_i, s_j, e_j, lda, 0, 0,
        chunk_size, a, b, c);
}

int main(int argc, char **argv){
    Framework::init();
    length = std::atoi(argv[1]);
    int max_run = std::atoi(argv[2]);

    Matrix* data1 = new Matrix(length,length);
    ArrayList* data2 = new ArrayList(length);
    ArrayList* data3 = new ArrayList(length);
    initialize(length, data1);
    initialize(data2, length);
    initialize(data3, length);

    double milliseconds = 0;
    for(int run = 0; run <= max_run; ++run){
        UserData_t* user = new UserData_t({data1}, {data2, data3});
        struct timeval start, end;
        gettimeofday(&start, NULL);
        parallel_for(new loopData_t(0, length, user), cfor_func, gfor_func);
        gettimeofday(&end, NULL);
        milliseconds += (end.tv_sec - start.tv_sec) * 1000 + 1.0e-3 * (end.tv_usec - start.tv_usec);
        delete user;
    }
    milliseconds /= max_run;
    std::cout << milliseconds << std::endl;
    delete data1;
    delete data2;
    delete data3;
    return 0;
}
