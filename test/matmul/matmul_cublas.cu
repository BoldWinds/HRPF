#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Helper function to check CUDA errors
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA Error: " << hipGetErrorString(error) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(1); \
        } \
    } while(0)

// Helper function to check cuBLAS errors
#define CHECK_CUBLAS_ERROR(call) \
    do { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            std::cerr << "cuBLAS Error: " << status << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(1); \
        } \
    } while(0)

double matmul_cublas(int n, int max_run) {
    double *A, *B, *C;
    
    // Allocate memory
    CHECK_CUDA_ERROR(hipMallocManaged(&A, n * n * sizeof(double)));
    CHECK_CUDA_ERROR(hipMallocManaged(&B, n * n * sizeof(double)));
    CHECK_CUDA_ERROR(hipMallocManaged(&C, n * n * sizeof(double)));

    // Initialize matrices
    for (int i = 0; i < n * n; i++) {
        A[i] = static_cast<double>(rand()) / RAND_MAX;
        B[i] = static_cast<double>(rand()) / RAND_MAX;
        C[i] = 0.0f;  // Initialize C to zeros
    }

    // Create cuBLAS handle
    hipblasHandle_t handle;
    CHECK_CUBLAS_ERROR(hipblasCreate(&handle));

    double alpha = 1.0f, beta = 0.0f;
    
    // Make sure data is ready on device before timing
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    auto start = std::chrono::high_resolution_clock::now();

    for (int run = 0; run < max_run; ++run) {
        CHECK_CUBLAS_ERROR(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, 
                                       &alpha, A, n, B, n, &beta, C, n));
    }
    
    // Ensure all GPU operations are completed before stopping the timer
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = end - start;

    // Clean up
    CHECK_CUBLAS_ERROR(hipblasDestroy(handle));
    CHECK_CUDA_ERROR(hipFree(A));
    CHECK_CUDA_ERROR(hipFree(B));
    CHECK_CUDA_ERROR(hipFree(C));

    return duration.count() / max_run;
}

int main(int argc, char* argv[]) {
    // Validate command line arguments
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <matrix_size> <max_run>" << std::endl;
        return 1;
    }
    
    int n, max_run;
    try {
        n = std::stoi(argv[1]);
        max_run = std::stoi(argv[2]);
        
        if (n <= 0 || max_run <= 0) {
            std::cerr << "Matrix size and repeat count must be positive integers" << std::endl;
            return 1;
        }
    } catch (const std::exception& e) {
        std::cerr << "Error parsing arguments: " << e.what() << std::endl;
        return 1;
    }
    
    double milliseconds = matmul_cublas(n, max_run);
    std::cout << milliseconds << std::endl;
    return 0;
}