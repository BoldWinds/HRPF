#include <iostream>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/execution_policy.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <random>


void test_sort(double* data_d, int len, hipStream_t stream, float& elapsed_time) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 记录排序开始时间
    hipEventRecord(start, stream);
    thrust::sort(thrust::cuda::par.on(stream), thrust::device_pointer_cast(data_d), thrust::device_pointer_cast(data_d + len));
    hipEventRecord(stop, stream);

    // 等待排序完成
    hipEventSynchronize(stop);

    // 获取排序所用的时间
    hipEventElapsedTime(&elapsed_time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dist(0.0, 10000.0);

    for (int n = 10485670; n <= 104857600; n += 10485670) {
        std::vector<double> data(n);
        for (int i = 0; i < n; ++i) {
            data[i] = dist(gen);
        }

        double* data_d;
        hipMalloc((void**)&data_d, n * sizeof(double));

        hipMemcpy(data_d, data.data(), n * sizeof(double), hipMemcpyHostToDevice);

        float elapsed_time = 0.0f;
        test_sort(data_d, n, stream, elapsed_time);

        std::cout << "Size: " << n << " elements, Time: " << elapsed_time << " ms" << std::endl;

        hipFree(data_d);
    }

    hipStreamDestroy(stream);
    return 0;
}
