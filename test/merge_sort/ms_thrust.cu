#include <iostream>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/execution_policy.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <random>


void test_sort(double* data_d, int len, hipStream_t stream, float& elapsed_time) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 记录排序开始时间
    hipEventRecord(start, stream);
    thrust::sort(thrust::cuda::par.on(stream), thrust::device_pointer_cast(data_d), thrust::device_pointer_cast(data_d + len));
    hipEventRecord(stop, stream);

    // 等待排序完成
    hipEventSynchronize(stop);

    // 获取排序所用的时间
    hipEventElapsedTime(&elapsed_time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main(int argc, char** argv){
    int n = std::atoi(argv[1]);
    int max_run = std::atoi(argv[2]);
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dist(0.0, 10000.0);

    double milliseconds = 0;
    for (int run = 0; run <= max_run; run++) {
        std::vector<double> data(n);
        for (int i = 0; i < n; ++i) {
            data[i] = dist(gen);
        }

        double* data_d;
        hipMalloc((void**)&data_d, n * sizeof(double));

        hipMemcpy(data_d, data.data(), n * sizeof(double), hipMemcpyHostToDevice);

        float elapsed_time = 0.0f;
        test_sort(data_d, n, stream, elapsed_time);
        milliseconds += elapsed_time;

        hipFree(data_d);
    }
    std::cout << milliseconds << std::endl;

    hipStreamDestroy(stream);
    return 0;
}
