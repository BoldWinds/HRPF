#include <iostream>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/execution_policy.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <random>

void test_sort(double* data_d, int len, hipStream_t stream, float& avg_time) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float total_time = 0.0f;

    for (int i = 0; i < 100; ++i) {  // 运行 100 次，取平均时间
        hipEventRecord(start, stream);
        thrust::sort(thrust::cuda::par.on(stream), thrust::device_pointer_cast(data_d), thrust::device_pointer_cast(data_d + len));
        hipEventRecord(stop, stream);
        hipEventSynchronize(stop);

        float elapsed_time;
        hipEventElapsedTime(&elapsed_time, start, stop);
        total_time += elapsed_time;
    }

    avg_time = total_time / 100;  // 计算平均时间

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dist(0.0, 10000.0);

    for (int n = 10'000'000; n <= 100'000'000; n += 10'000'000) { // 10M 到 100M，步长 10M
        std::vector<double> data(n);
        for (int i = 0; i < n; ++i) {
            data[i] = dist(gen);
        }

        double* data_d;
        hipMalloc((void**)&data_d, n * sizeof(double));

        hipMemcpy(data_d, data.data(), n * sizeof(double), hipMemcpyHostToDevice);

        float avg_time = 0.0f;
        test_sort(data_d, n, stream, avg_time);

        std::cout << "Size: " << n << " elements, Avg Time: " << avg_time << " ms" << std::endl;

        hipFree(data_d);
    }

    hipStreamDestroy(stream);
    return 0;
}
