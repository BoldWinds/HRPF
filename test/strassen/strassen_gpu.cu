#include "strassen.h"
#define THRESHOLD 256  // Increased threshold for GPU implementation

class CudaMatrix : public BasicMatrix {
public:
    CudaMatrix(int dim) : BasicMatrix(dim) {
        copyToGPU();
    }

    ~CudaMatrix() {}

    void matrixAdd(const BasicMatrix& A, const BasicMatrix& B, BasicMatrix& C) override {
        int n = A.getDim();
        
        const double* a = A.getGPUData();
        const double* b = B.getGPUData();
        double* c = C.getGPUData();
        
        // Use the provided CUDA function for matrix addition
        hipStream_t stream;
        hipStreamCreate(&stream);
        sumMatrix((double*)a, (double*)b, c, n, n, n, n, stream);
        hipStreamSynchronize(stream);
        hipStreamDestroy(stream);
    }

    void matrixSub(const BasicMatrix& A, const BasicMatrix& B, BasicMatrix& C) override {
        int n = A.getDim();
        
        const double* a = A.getGPUData();
        const double* b = B.getGPUData();
        double* c = C.getGPUData();
        
        // Use the provided CUDA function for matrix subtraction
        hipStream_t stream;
        hipStreamCreate(&stream);
        subMatrix((double*)a, (double*)b, c, n, n, n, n, stream);
        hipStreamSynchronize(stream);
        hipStreamDestroy(stream);
    }

    void matrixMul(const BasicMatrix& A, const BasicMatrix& B, BasicMatrix& C) override {
        int n = A.getDim();
        
        const double* a = A.getGPUData();
        const double* b = B.getGPUData();
        double* c = C.getGPUData();
        
        // Use the provided CUDA function for matrix multiplication
        hipStream_t stream;
        hipStreamCreate(&stream);
        gemm((double*)a, (double*)b, c, n, n, n, n, stream);
        hipStreamSynchronize(stream);
        hipStreamDestroy(stream);
    }

    void splitMatrix(const BasicMatrix& A, BasicMatrix& A11, BasicMatrix& A12, 
                     BasicMatrix& A21, BasicMatrix& A22) override {
        int n = A.getDim();
        int newSize = n / 2;
        
        // First copy matrices from GPU to CPU for easier splitting
        // (This could be optimized with a custom CUDA kernel for better performance)
        const double* src = A.getData();
        
        double* a11 = A11.getData();
        double* a12 = A12.getData();
        double* a21 = A21.getData();
        double* a22 = A22.getData();
        
        #pragma omp parallel for
        for (int i = 0; i < newSize; i++) {
            for (int j = 0; j < newSize; j++) {
                a11[i * newSize + j] = src[i * n + j];
                a12[i * newSize + j] = src[i * n + j + newSize];
                a21[i * newSize + j] = src[(i + newSize) * n + j];
                a22[i * newSize + j] = src[(i + newSize) * n + j + newSize];
            }
        }
        
        // Copy back to GPU
        A11.copyToGPU();
        A12.copyToGPU();
        A21.copyToGPU();
        A22.copyToGPU();
    }

    void mergeMatrix(BasicMatrix& C, const BasicMatrix& C11, const BasicMatrix& C12,
                     const BasicMatrix& C21, const BasicMatrix& C22) override {
        int n = C.getDim();
        int oldSize = n / 2;
        
        // First, copy matrices from GPU to CPU for easier merging
        // (This could be optimized with a custom CUDA kernel for better performance)
        double* dst = C.getData();
        
        const double* c11 = C11.getData();
        const double* c12 = C12.getData();
        const double* c21 = C21.getData();
        const double* c22 = C22.getData();
        
        #pragma omp parallel for
        for (int i = 0; i < oldSize; i++) {
            for (int j = 0; j < oldSize; j++) {
                dst[i * n + j] = c11[i * oldSize + j];
                dst[i * n + j + oldSize] = c12[i * oldSize + j];
                dst[(i + oldSize) * n + j] = c21[i * oldSize + j];
                dst[(i + oldSize) * n + j + oldSize] = c22[i * oldSize + j];
            }
        }
        
        // Copy back to GPU
        C.copyToGPU();
    }

    void strassenMultiply(const BasicMatrix& A, const BasicMatrix& B, BasicMatrix& C) {
        int n = A.getDim();
        
        if (n <= THRESHOLD) {
            matrixMul(A, B, C);
            return;
        }
        
        int newSize = n / 2;
        
        // Create submatrices
        CudaMatrix A11(newSize), A12(newSize), A21(newSize), A22(newSize);
        CudaMatrix B11(newSize), B12(newSize), B21(newSize), B22(newSize);
        CudaMatrix C11(newSize), C12(newSize), C21(newSize), C22(newSize);
        
        // Split input matrices
        splitMatrix(A, A11, A12, A21, A22);
        splitMatrix(B, B11, B12, B21, B22);
        
        // Temporary matrices for intermediate results
        CudaMatrix M1(newSize), M2(newSize), M3(newSize), M4(newSize);
        CudaMatrix M5(newSize), M6(newSize), M7(newSize);
        
        // Temporary matrices for additions/subtractions
        CudaMatrix temp1(newSize), temp2(newSize);
        
        // Create CUDA streams for potential parallelism
        hipStream_t streams[7];
        for (int i = 0; i < 7; i++) {
            hipStreamCreate(&streams[i]);
        }
        
        // M1 = (A11 + A22) * (B11 + B22)
        matrixAdd(A11, A22, temp1);
        matrixAdd(B11, B22, temp2);
        strassenMultiply(temp1, temp2, M1);
        
        // M2 = (A21 + A22) * B11
        matrixAdd(A21, A22, temp1);
        strassenMultiply(temp1, B11, M2);
        
        // M3 = A11 * (B12 - B22)
        matrixSub(B12, B22, temp1);
        strassenMultiply(A11, temp1, M3);
        
        // M4 = A22 * (B21 - B11)
        matrixSub(B21, B11, temp1);
        strassenMultiply(A22, temp1, M4);
        
        // M5 = (A11 + A12) * B22
        matrixAdd(A11, A12, temp1);
        strassenMultiply(temp1, B22, M5);
        
        // M6 = (A21 - A11) * (B11 + B12)
        matrixSub(A21, A11, temp1);
        matrixAdd(B11, B12, temp2);
        strassenMultiply(temp1, temp2, M6);
        
        // M7 = (A12 - A22) * (B21 + B22)
        matrixSub(A12, A22, temp1);
        matrixAdd(B21, B22, temp2);
        strassenMultiply(temp1, temp2, M7);
        
        // Synchronize all streams
        for (int i = 0; i < 7; i++) {
            hipStreamSynchronize(streams[i]);
        }
        
        // Calculate C11, C12, C21, C22
        // C11 = M1 + M4 - M5 + M7
        matrixAdd(M1, M4, temp1);
        matrixSub(temp1, M5, temp2);
        matrixAdd(temp2, M7, C11);
        
        // C12 = M3 + M5
        matrixAdd(M3, M5, C12);
        
        // C21 = M2 + M4
        matrixAdd(M2, M4, C21);
        
        // C22 = M1 + M3 - M2 + M6
        matrixAdd(M1, M3, temp1);
        matrixSub(temp1, M2, temp2);
        matrixAdd(temp2, M6, C22);
        
        // Merge results into C
        mergeMatrix(C, C11, C12, C21, C22);
        
        // Destroy streams
        for (int i = 0; i < 7; i++) {
            hipStreamDestroy(streams[i]);
        }
    }
};

class CudaStrassen : public Strassen {
public:
    CudaStrassen(int dim) : Strassen(dim) {
        A = new CudaMatrix(dim);
        B = new CudaMatrix(dim);
        C = new CudaMatrix(dim);
    }

    ~CudaStrassen() {}

    void prepare() override {
        A->generateRandomMatrix();
        B->generateRandomMatrix();
        A->copyToGPU();
        B->copyToGPU();
    }

    // Strassen matrix multiplication implementation
    void run() override {
        static_cast<CudaMatrix*>(C)->strassenMultiply(*A, *B, *C);
        C->copyFromGPU();  // Get the result back to CPU memory
    }
};

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <matrix_size> <number_of_runs>\n";
        return 1;
    }
    
    int n = std::atoi(argv[1]);
    int max_run = std::atoi(argv[2]);
    
    hipFree(0);
    
    CudaStrassen strassen(n);
    double milliseconds = strassen.test(max_run);
    std::cout << milliseconds << std::endl;
    
    return 0;
}